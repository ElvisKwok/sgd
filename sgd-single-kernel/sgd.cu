#include "hip/hip_runtime.h"
#include "sgd.h"

// unused code
/*
__global__ void addKernel_3(int *c, const int *a, const int *b, unsigned int size)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x; // threadIdx.x�� ĳ��block�ڵ��߳��±�
	int stride = blockDim.x * gridDim.x;	// blockDim.x��1��block�ж��ٸ�thread, gridDim.x��1��grid�ж��ٸ�block
	for (int i = tid; i < size; i += stride)
	{
		c[tid] = a[tid] + b[tid];
	}
}


void solveByGPU(int *a, int *b, int *c, int size)
{
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	dim3 dim_grid, dim_block;
	dim_block.x = min(prop.maxThreadsDim[0], prop.maxThreadsPerBlock);
	if (dim_block.x >= size) {
		dim_block.x = size;
	}
	dim_grid.x = size / dim_block.x;
	if (size % dim_block.x != 0) {
		dim_grid.x++;
	}


	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipMalloc((void**)&dev_a, size * sizeof(int));
	hipMalloc((void**)&dev_b, size * sizeof(int));
	hipMalloc((void**)&dev_c, size * sizeof(int));
	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

	addKernel_3 <<< dim_grid, dim_block >> >(dev_c, dev_a, dev_b, size);

	hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}

*/


// TO-DO:
// �����ڻ�
__device__ typeRate innerProduct(
								 int userIdx,
								 int itemIdx,
								 typeRate **matrixUser,
								 typeRate **matrixItem,
								 int K						// ��������ά��
								)
{
	typeRate predictRate = 0;
	for (int k = 0; k < K; ++k)
	{
		predictRate += matrixUser[userIdx][k] * matrixItem[itemIdx][k];
	}
	return predictRate;
}


// sgdһ�θ��²���
__device__ void sgdUpdate(
						  typeRate rate,
						  int userIdx,
						  int itemIdx,
						  typeRate **matrixUser,
						  typeRate **matrixItem,
						  int K,					// ��������ά��
						  double lambda,			// ����ϵ��
						  double gamma				// ѧϰ��
						 )
{
	typeRate predictRate = innerProduct(userIdx, itemIdx, matrixUser, matrixItem, K);
	typeRate error = rate - predictRate;
	for (int k = 0; k < K; ++k)
	{
		matrixUser[userIdx][k] += (gamma * (2 * error * matrixItem[itemIdx][k] - lambda * matrixUser[userIdx][k]));
		matrixItem[itemIdx][k] += (gamma * (2 * error * matrixUser[userIdx][k] - lambda * matrixItem[itemIdx][k]));
	}
}


__global__ void sgd_kernel(
							sRateNode *rateNodeArray,
							typeRate **d_matrixUser,
							typeRate **d_matrixItem,
							sWorkset *d_worksetArray,
							sWorkseg **d_mWorkseg,
							int **d_matrixPattern,
							int s,						// ��s��ģʽ
							int subBlockNumL,			// subBlockNumL * subBlockNumL���ӿ� 
							int subBlockLen,			// �ӿ��СΪ subBlockLen * subBlockLen
							int K,						// ��������ά��
							double lambda,				// ����ϵ��
							double gamma				// ѧϰ��
						   )
{
	int bidx = blockIdx.x;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	// TO-DO: �ж�bidxԽ�磿 subBlockNumL
	if (bidx > subBlockNumL)
	{
		return;
	}
	int bid = d_matrixPattern[s][bidx];
	
	if (d_worksetArray[bid].beg == d_worksetArray[bid].end)
	{
		return;
	}

	for (int tag = 0; tag < subBlockLen; ++tag)
	{
		int from = d_mWorkseg[bid][tag].from;
		int to = d_mWorkseg[bid][tag].to;
		for (int iRate = from + tid; iRate < to; iRate += blockDim.x)	// iRateΪ �����ӿ�bid && ��ǩΪtag ������ֵ ��rateNodeArray������±�
		{
			typeRate rate = rateNodeArray[iRate].rate;
			int userIdx = rateNodeArray[iRate].u;
			int itemIdx = rateNodeArray[iRate].i;
			sgdUpdate(rate, userIdx, itemIdx, d_matrixUser, d_matrixItem, K, lambda, gamma);
		}

		// wait for all threads in this block to arrive here(i.e. current tag finish)
		__syncthreads();
	}

}


void solveByGPU(int *a, int *b, int *c, int size)
{
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	dim3 dim_grid, dim_block;
	dim_block.x = min(prop.maxThreadsDim[0], prop.maxThreadsPerBlock);
	if (dim_block.x >= size) {
		dim_block.x = size;
	}
	dim_grid.x = size / dim_block.x;
	if (size % dim_block.x != 0) {
		dim_grid.x++;
	}


	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipMalloc((void**)&dev_a, size * sizeof(int));
	hipMalloc((void**)&dev_b, size * sizeof(int));
	hipMalloc((void**)&dev_c, size * sizeof(int));
	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

	/*
	sgd_kernel << < dim_grid, dim_block >> >(dev_c, dev_a, dev_b, size);
	*/

	hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}