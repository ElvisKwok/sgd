#include "hip/hip_runtime.h"
#include "sgd.h"
#define CHECK(res) if(res!=hipSuccess){exit(-1);} 


// unused code
/*
__global__ void addKernel_3(int *c, const int *a, const int *b, unsigned int size)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x; // threadIdx.x�� ĳ��block�ڵ��߳��±�
	int stride = blockDim.x * gridDim.x;	// blockDim.x��1��block�ж��ٸ�thread, gridDim.x��1��grid�ж��ٸ�block
	for (int i = tid; i < size; i += stride)
	{
		c[tid] = a[tid] + b[tid];
	}
}


void solveByGPU(int *a, int *b, int *c, int size)
{
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	dim3 dim_grid, dim_block;
	dim_block.x = min(prop.maxThreadsDim[0], prop.maxThreadsPerBlock);
	if (dim_block.x >= size) {
		dim_block.x = size;
	}
	dim_grid.x = size / dim_block.x;
	if (size % dim_block.x != 0) {
		dim_grid.x++;
	}


	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipMalloc((void**)&dev_a, size * sizeof(int));
	hipMalloc((void**)&dev_b, size * sizeof(int));
	hipMalloc((void**)&dev_c, size * sizeof(int));
	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

	addKernel_3 <<< dim_grid, dim_block >> >(dev_c, dev_a, dev_b, size);

	hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}

*/


// TO-DO:
// �����ڻ�
__device__ typeRate innerProduct(
								 int userIdx,
								 int itemIdx,
								 typeRate *matrixUser,
								 typeRate *matrixItem,
								 int K						// ��������ά��
								)
{
	typeRate predictRate = 0;
	for (int k = 0; k < K; ++k)
	{
		predictRate += (*(matrixUser + userIdx*K +k)) * (*(matrixItem + itemIdx*K+k));
	}
	return predictRate;
}


// sgdһ�θ��²���
__device__ void sgdUpdate(
						  typeRate rate,
						  int userIdx,
						  int itemIdx,
						  typeRate *matrixUser,
						  typeRate *matrixItem,
						  int K,					// ��������ά��
						  double lambda,			// ����ϵ��
						  double gamma				// ѧϰ��
						 )
{
	typeRate predictRate = innerProduct(userIdx, itemIdx, matrixUser, matrixItem, K);
	typeRate error = rate - predictRate;
	typeRate tmp;
	for (int k = 0; k < K; ++k)
	{
		/*
		(*(matrixUser + userIdx*K + k)) += (gamma * (2 * error * (*(matrixItem + itemIdx*K + k)) - lambda * (*(matrixUser + userIdx*K + k))));
		(*(matrixItem + itemIdx*K + k)) += (gamma * (2 * error * (*(matrixUser + userIdx*K + k)) - lambda * (*(matrixItem + itemIdx*K + k))));
		*/
		tmp = (*(matrixUser + userIdx*K + k));
		(*(matrixUser + userIdx*K + k)) += (gamma * (error * (*(matrixItem + itemIdx*K + k)) - lambda * tmp));
		(*(matrixItem + itemIdx*K + k)) += (gamma * (error * tmp - lambda * (*(matrixItem + itemIdx*K + k))));
	}
}


__global__ void sgd_kernel(
							sRateNode *d_rateNodeArray,
							typeRate *d_matrixUser,
							typeRate *d_matrixItem,
							sWorkset *d_worksetArray,
							sWorkseg *d_mWorkseg,
							int *d_matrixPattern,
							int s,						// ��s��ģʽ
							int subBlockNumL,			// subBlockNumL * subBlockNumL���ӿ� 
							int subBlockLen,			// �ӿ��СΪ subBlockLen * subBlockLen
							int K,						// ��������ά��
							double lambda,				// ����ϵ��
							double gamma				// ѧϰ��
						   )
{
	int bidx = blockIdx.x;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	// TO-DO: �ж�bidxԽ�磿 subBlockNumL
	if (bidx > subBlockNumL)
	{
		return;
	}
	int bid = *(d_matrixPattern + s*subBlockNumL + bidx);
	
	// c++��� endΪ��һ��λ��
	if ((bid == -1) || d_worksetArray[bid].beg == d_worksetArray[bid].end)	// �տ��patternĬ�ϳ�ʼ��Ϊ-1
	{
		return;
	}

	for (int tag = 0; tag < subBlockLen; ++tag)
	{
		int from = (*(d_mWorkseg + bid*subBlockLen + tag)).from;
		int to = (*(d_mWorkseg + bid*subBlockLen + tag)).to;
		// QUESTION: tid��0��ʼ��
		// c++��� endΪ��һ��λ��
		for (int iRate = from + tid; iRate < to; iRate += blockDim.x)	// iRateΪ �����ӿ�bid && ��ǩΪtag ������ֵ ��rateNodeArray������±�
		{
			typeRate rate = d_rateNodeArray[iRate].rate;
			int userIdx = d_rateNodeArray[iRate].u - 1;
			int itemIdx = d_rateNodeArray[iRate].i - 1;
			sgdUpdate(rate, userIdx, itemIdx, d_matrixUser, d_matrixItem, K, lambda, gamma);
			printf("userIdx = %d, itemIdx = %d\n", userIdx, itemIdx);
		}

		// wait for all threads in this block to arrive here(i.e. current tag finish)
		__syncthreads();
	}

}


typeRate computeRMSE(sRateNode *rateNodeArray, typeRate *matrixUser,typeRate *matrixItem, int NNZ)
{
	int userIdx;
	int itemIdx;
	typeRate predictRate = 0.0;
	typeRate err_sum = 0.0;
	for (int i = 0; i < NNZ; ++i)
	{
		userIdx = rateNodeArray[i].u - 1;
		itemIdx = rateNodeArray[i].i - 1;
		predictRate = innerProduct(matrixUser, matrixItem, userIdx, itemIdx);
		//cout << "(" << userIdx+1 << ", " << itemIdx+1 << "): "<<predictRate << endl;
		err_sum += pow((rateNodeArray[i].rate - predictRate), 2);
	}
	return sqrt(err_sum / NNZ);
}

void solveByGPU(
				sRateNode *rateNodeArray,
				typeRate *matrixUser,
				typeRate *matrixItem,
				sWorkset *worksetArray,
				sWorkseg *mWorkseg,
				int *matrixPattern,
				int subBlockNumL,			// subBlockNumL * subBlockNumL���ӿ� 
				int subBlockLen,			// �ӿ��СΪ subBlockLen * subBlockLen
				double lambda,				// ����ϵ��
				double gamma,				// ѧϰ��
				int NNZ						// ����ֵ����
			   )
{
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	// QUESTION: dim_grid.x(blockά��)��dim_block.x(threadά��)���ã�
	dim3 dim_grid, dim_block;
	dim_block.x = min(prop.maxThreadsDim[0], prop.maxThreadsPerBlock);
	/*
	if (dim_block.x >= size) {	
		dim_block.x = size;
	}
	dim_grid.x = size / dim_block.x;
	if (size % dim_block.x != 0) {
		dim_grid.x++;
	}
	*/
	dim_grid.x = subBlockNumL;


	sRateNode *d_rateNodeArray;
	typeRate *d_matrixUser;
	typeRate *d_matrixItem;
	sWorkset *d_worksetArray;
	sWorkseg *d_mWorkseg;
	int *d_matrixPattern;
	
	int subBlockNum = subBlockNumL*subBlockNumL;

	hipError_t res;
	res = hipMalloc((void**)(&d_rateNodeArray), NNZ * sizeof(sRateNode)); CHECK(res)
	res = hipMalloc((void**)(&d_matrixUser), M*K * sizeof(typeRate)); CHECK(res)
	res = hipMalloc((void**)(&d_matrixItem), N*K * sizeof(typeRate)); CHECK(res)
	res = hipMalloc((void**)(&d_worksetArray), subBlockNum * sizeof(sWorkset)); CHECK(res)
	res = hipMalloc((void**)(&d_mWorkseg), subBlockNum*subBlockLen * sizeof(sWorkseg)); CHECK(res)
	res = hipMalloc((void**)(&d_matrixPattern), subBlockNum * sizeof(int)); CHECK(res)


	res = hipMemcpy(d_rateNodeArray, rateNodeArray, NNZ * sizeof(sRateNode), hipMemcpyHostToDevice); CHECK(res)
	res = hipMemcpy(d_matrixUser, matrixUser, M*K * sizeof(typeRate), hipMemcpyHostToDevice); CHECK(res)
	res = hipMemcpy(d_matrixItem, matrixItem, N*K * sizeof(typeRate), hipMemcpyHostToDevice); CHECK(res)
	res = hipMemcpy(d_worksetArray, worksetArray, subBlockNum * sizeof(sWorkset), hipMemcpyHostToDevice); CHECK(res)
	res = hipMemcpy(d_mWorkseg, mWorkseg, subBlockNum*subBlockLen * sizeof(sWorkseg), hipMemcpyHostToDevice); CHECK(res)
	res = hipMemcpy(d_matrixPattern, matrixPattern, subBlockNum * sizeof(int), hipMemcpyHostToDevice); CHECK(res)

		/*
		sgd_kernel << < dim_grid, dim_block >> >(dev_c, dev_a, dev_b, size);
		*/
		for (int iter = 0; iter < MAX_ITER; ++iter)
		{
			for (int s = 0; s < subBlockNumL; ++s)
			{
				sgd_kernel <<< dim_grid, dim_block >>> (
															d_rateNodeArray,
															d_matrixUser,
															d_matrixItem,
															d_worksetArray,
															d_mWorkseg,
															d_matrixPattern,
															s,						// ��s��ģʽ
															subBlockNumL,			// subBlockNumL * subBlockNumL���ӿ� 
															subBlockLen,			// �ӿ��СΪ subBlockLen * subBlockLen
															K,						// ��������ά��
															lambda,					// ����ϵ��
															gamma					// ѧϰ��
														);
			}
			//hipMemcpy(matrixUser, d_matrixUser, M*K * sizeof(typeRate), hipMemcpyDeviceToHost); CHECK(res)
			//printMatrix(matrixUser, M, K);
		}
	
	res = hipMemcpy(matrixUser, d_matrixUser, M*K * sizeof(typeRate), hipMemcpyDeviceToHost); CHECK(res)
	res = hipMemcpy(matrixItem, d_matrixItem, N*K * sizeof(typeRate), hipMemcpyDeviceToHost); CHECK(res)

	cout << "RMSE: " << computeRMSE(rateNodeArray, matrixUser, matrixItem, NNZ) << endl;

	hipFree(d_rateNodeArray);
	hipFree(d_matrixUser);
	hipFree(d_matrixItem);
	hipFree(d_worksetArray);
	hipFree(d_mWorkseg);
	hipFree(d_matrixPattern);
}